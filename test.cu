#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void forward_kernel(const float* Q,const float* K,const float* V,const int N,const int d,const int Tc,const int Tr, const int Bc,const int Br,const float scale,float* l,float* m,float* O){
    int tid = threadIdx.x;
    int bid_x = blockIdx.x;
    int bid_y = blockIdx.y;

    int qkv_offset = (bid_x * gridDim.x * N * d) + (bid_y * N * d);
    int lm_offset = (bid_x * gridDim.x * N) + (bid_y * N);


    extern __shared__ float sram[];

    int tile_size = Bc*d;

    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size*2];
    float* S = &sram[tile_size*3];

    for(int j=0;j<Tc;j++){
        for(int x=0;x++;x<d){
            Kj[(tid*d)+x] = K[qkv_offset + (tile_size*j) + (tid*d)+x];
            Vj[(tid*d)+x] = V[qkv_offset + (tile_size*j) + (tid*d)+x];
        }

        __syncthreads();

        for(int i=0;i<Tr;i++){
            for(int x=0;x++;x<d){
            Qi[(tid*d)+x] = Q[qkv_offset + (tile_size*j) + (tid*d)+x];
        }
        float row_m_prev = m[lm_offset + (Br*i) + tid];
        float row_l_prev = l[lm_offset + (Br*i) + tid];

        float row_m = -INFINITY;
        for(int y=0;y<Bc;y++){
            float sum = 0;
            for(int x = 0;x<Br;x++){
                sum+=  Qi[(tid*d)+x] * Kj[(tid*d)+y];
            }
        }
        }
    }



}