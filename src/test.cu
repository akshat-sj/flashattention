#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void forward_kernel_v1(const float* Q,const float* K,const float* V,const int N,const int d,const int Tc,const int Tr, const int Bc,const int Br,const float scale,float* l,float* m,float* O){
    int tid = threadIdx.x;
    int bid_x = blockIdx.x;
    int bid_y = blockIdx.y;

    int qkv_offset = (bid_x * gridDim.y * N * d) + (bid_y * N * d);
    int lm_offset = (bid_x * gridDim.y * N) + (bid_y * N);


    extern __shared__ float sram[];

    int tile_size = Bc*d;

    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size*2];
    float* S = &sram[tile_size*3];

    for(int j=0;j<Tc;j++){
        for(int x=0;x<d;x++){
            Kj[(tid*d)+x] = K[qkv_offset + (tile_size*j) + (tid*d)+x];
            Vj[(tid*d)+x] = V[qkv_offset + (tile_size*j) + (tid*d)+x];
        }

        __syncthreads();

        for(int i=0;i<Tr;i++){
            for(int x=0;x<d;x++){
            Qi[(tid*d)+x] = Q[qkv_offset + (tile_size*i) + (tid*d)+x];
        }
        float row_m_prev = m[lm_offset + (Br*i) + tid];
        float row_l_prev = l[lm_offset + (Br*i) + tid];

        float row_m = -INFINITY;
        for(int y=0;y<Bc;y++){
            float sum = 0;
            for(int x = 0;x<d;x++){
                sum+=  Qi[(tid*d)+x] * Kj[(y*d)+x];
            }
            sum*=scale;
            S[(Bc*tid) + y] = sum;
            if(sum>row_m){
                row_m = sum;
                }
            }
        float row_l = 0;
        for(int y =0;y<Bc;y++){
            S[(Bc * tid) + y] = __expf(S[(Bc * tid) + y] - row_m);
            row_l +=  S[(Bc * tid) + y];
            }
        
        float row_m_new = max(row_m_prev,row_m);
        float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

        for(int x = 0; x < d;x++){
                float pv = 0;
                for(int y=0;y<Bc;y++){
                    pv += S[(Bc * tid) + y] * Vj[(y*d)+x];
                }
                O[qkv_offset + (tile_size*i) + (tid*d)+x] = (1/(row_l_new)) * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size*i) + (tid*d)+x]) + (__expf(row_m - row_m_new)*pv));
            }
            m[lm_offset + (Br*i) + tid] = row_m_new;
            l[lm_offset + (Br*i) + tid] = row_l_new;
        }

        __syncthreads();
    }

}



__global__ void forward_kernel_v2(const float* Q, const float* K, const float* V, int N, int d, int Tc, int Tr, int Bc, int Br, float scale, float* O, float* L) {
    int tid = threadIdx.x;
    int bid_x = blockIdx.x;
    int bid_y = blockIdx.y;

    int qkv_offset = (bid_x * gridDim.y * N * d) + (bid_y * N * d);

    extern __shared__ float sram[];

    int tile_size = Bc * d;

    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[2 * tile_size];
    float* S = &sram[3 * tile_size];

    for (int i = 0; i < Tr; i++) {
        for (int x = 0; x < d; x++) {
            Qi[(tid * d) + x] = Q[qkv_offset + (tile_size * i) + (tid * d) + x];
        }

        __syncthreads();

        float row_m_prev = -INFINITY;
        float row_l_prev = 0.0f;
        float row_m_new ;
        float row_l_new;

        for (int j = 0; j < Tc; j++) {
            for (int x = 0; x < d; x++) {
                Kj[(tid * d) + x] = K[qkv_offset + (tile_size * j) + (tid * d) + x];
                Vj[(tid * d) + x] = V[qkv_offset + (tile_size * j) + (tid * d) + x];
            }

            float row_m = -INFINITY;

            for (int y = 0; y < Bc; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tid * d) + x] * Kj[(y * d) + x];
                }
                sum *= scale;
                S[(Bc * tid) + y] = sum;
                if (sum > row_m) {
                    row_m = sum;
                }
            }

            row_m_new = max(row_m_prev, row_m);

            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                S[(Bc * tid) + y] = __expf(S[(Bc * tid) + y] - row_m_new);
                row_l += S[(Bc * tid) + y];
            }

            row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + row_l;

            for (int x = 0; x < d; x++) {
                float pv = 0;
                for (int y = 0; y < Bc; y++) {
                    pv += S[(Bc * tid) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tid * d) + x] = (__expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tid * d) + x] + pv);
            }

            row_m_prev = row_m_new;
            row_l_prev = row_l_new;
        }

        __syncthreads();

        for (int x = 0; x < d; x++) {
            O[qkv_offset + (tile_size * i) + (tid * d) + x] *= 1 / row_l_prev;
        }
        L[qkv_offset + (tile_size * i) + (tid * d)] = row_m_new + __logf(row_l_new);
    }
}

__global__ void forward_kernel_optimized(const float* __restrict__ Q, const float* __restrict__ K, const float* __restrict__ V, int N, int d, int Tc, int Tr, int Bc, int Br, float scale, float* O) {
    const int tid = threadIdx.x;
    const int bid_x = blockIdx.x;
    const int bid_y = blockIdx.y;
    const int qkv_offset = (bid_x * gridDim.y * N * d) + (bid_y * N * d);
    
    extern __shared__ float sram[];
    const int tile_size = Bc * d;
    
    float* __restrict__ Qi = sram;
    float* __restrict__ Kj = &sram[tile_size];
    float* __restrict__ Vj = &sram[2 * tile_size];
    float* __restrict__ S = &sram[3 * tile_size];

    constexpr int CHUNK_SIZE = 32;  

    for (int i = 0; i < Tr; i++) {
        #pragma unroll
        for (int x = 0; x < d; x += CHUNK_SIZE) {
            #pragma unroll 1
            for (int k = 0; k < CHUNK_SIZE && (x + k) < d; k++) {
                Qi[(tid * d) + x + k] = __ldg(&Q[qkv_offset + (tile_size * i) + (tid * d) + x + k]);
            }
        }
        __syncthreads();

        float row_m_prev = -INFINITY;
        float row_l_prev = 0.0f;

        for (int j = 0; j < Tc; j++) {
            #pragma unroll
            for (int x = 0; x < d; x += CHUNK_SIZE) {
                #pragma unroll 1
                for (int k = 0; k < CHUNK_SIZE && (x + k) < d; k++) {
                    Kj[(tid * d) + x + k] = __ldg(&K[qkv_offset + (tile_size * j) + (tid * d) + x + k]);
                    Vj[(tid * d) + x + k] = __ldg(&V[qkv_offset + (tile_size * j) + (tid * d) + x + k]);
                }
            }
            __syncthreads();

            float row_m = -INFINITY;
            
            #pragma unroll
            for (int y = 0; y < Bc; y++) {
                float sum = 0.0f;
                #pragma unroll
                for (int x = 0; x < d; x += CHUNK_SIZE) {
                    float partial_sum = 0.0f;
                    #pragma unroll 1
                    for (int k = 0; k < CHUNK_SIZE && (x + k) < d; k++) {
                        partial_sum = fmaf(Qi[(tid * d) + x + k], Kj[(y * d) + x + k], partial_sum);
                    }
                    sum += partial_sum;
                }
                sum *= scale;
                S[(Bc * tid) + y] = sum;
                row_m = fmaxf(row_m, sum);
            }
            
            float row_m_new = fmaxf(row_m_prev, row_m);
            float row_l = 0.0f;

            #pragma unroll 1
            for (int y = 0; y < Bc; y++) {
                S[(Bc * tid) + y] = __expf(S[(Bc * tid) + y] - row_m_new);
                row_l += S[(Bc * tid) + y];
            }

            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + row_l;

            #pragma unroll
            for (int x = 0; x < d; x += CHUNK_SIZE) {
                #pragma unroll 1
                for (int k = 0; k < CHUNK_SIZE && (x + k) < d; k++) {
                    float pv = 0.0f;
                    #pragma unroll
                    for (int y = 0; y < Bc; y++) {
                        pv = fmaf(S[(Bc * tid) + y], Vj[(y * d) + x + k], pv);
                    }
                    const int out_idx = qkv_offset + (tile_size * i) + (tid * d) + x + k;
                    float prev_val = __ldg(&O[out_idx]);
                    O[out_idx] = fmaf(__expf(row_m_prev - row_m_new), prev_val, pv);
                }
            }
            
            row_m_prev = row_m_new;
            row_l_prev = row_l_new;
            __syncthreads();
        }

        const float inv_row_l = 1.0f / row_l_prev;
        #pragma unroll
        for (int x = 0; x < d; x += CHUNK_SIZE) {
            #pragma unroll 1
            for (int k = 0; k < CHUNK_SIZE && (x + k) < d; k++) {
                const int out_idx = qkv_offset + (tile_size * i) + (tid * d) + x + k;
                O[out_idx] *= inv_row_l;
            }
        }
        __syncthreads();
    }
}


torch::Tensor forward(torch::Tensor Q,torch::Tensor K, torch::Tensor V){
    const int Bc = 32;
    const int Br = 32;

    const int B = Q.size(0);
    const int nh = Q.size(1);
    const int N = Q.size(2);
    const int d = Q.size(3);

    const int Tc = ceil((float) N/ Bc);
    const int Tr = ceil((float) N/ Br);

    const float scale  = 1.0/sqrt(d);

    auto O = torch::zeros_like(Q);
    auto l = torch::zeros({B,nh,N});
    auto L = torch::zeros_like(Q);
    auto m = torch::full({B,nh,N},-INFINITY);
    torch::Device device(torch::kCUDA);
    l = l.to(device);
    m = m.to(device);

    const int sram_size = (4*Bc*d + Bc*Br) * sizeof(float);
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \\n", max_sram_size, sram_size);

    dim3 grid_dim(B,nh);
    dim3 block_dim(Bc);

    forward_kernel_optimized<<<grid_dim,block_dim,sram_size>>>(Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(), N, d, Tc, Tr, Bc, Br, scale, O.data_ptr<float>());

return O;

}